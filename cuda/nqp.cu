#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Solves N-Queens iteratively using bitwise operations
__device__ void solveNQueensIterativeGPU(int rowMask, int ldMask, int rdMask, int n, int *localCount) {
    int allRows = (1 << n) - 1;
    int stack[100];  // stack to simulate recursion, max depth = 100
    int sp = 0;     // stack pointer
    int safe, p;

    stack[sp++] = rowMask;
    stack[sp++] = ldMask;
    stack[sp++] = rdMask;

    while (sp > 0) {
        rdMask = stack[--sp];
        ldMask = stack[--sp];
        rowMask = stack[--sp];

        if (rowMask == allRows) {
            (*localCount)++;  // Found a solution
            continue;
        }

        safe = allRows & (~(rowMask | ldMask | rdMask));
        while (safe) {
            p = safe & (-safe);  // Least significant bit
            safe -= p;
            stack[sp++] = rowMask | p;
            stack[sp++] = (ldMask | p) << 1;
            stack[sp++] = (rdMask | p) >> 1;
        }
    }
}

__global__ void nQueensKernel(int n, int *globalCount) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int rowMask = 1 << idx;
    int ldMask = rowMask << 1;
    int rdMask = rowMask >> 1;

    if (idx >= n) return;

    int localCount = 0;  // Local counter for each thread
    solveNQueensIterativeGPU(rowMask, ldMask, rdMask, n, &localCount);

    atomicAdd(globalCount, localCount);  // Aggregate results to global count
}

int main() {
    std::cout << "N       Number of Solutions     Execution Time (seconds)" << std::endl;

    // Loop through N = 1 to 15 (excluding N = 16)
    for (int n = 1; n <= 16; n++) {
        int *d_count;
        hipMalloc(&d_count, sizeof(int));
        hipMemset(d_count, 0, sizeof(int));  // Initialize count to 0

        auto start = std::chrono::high_resolution_clock::now();

        int blockSize = 1000;  // Adjust block size depending on N
        int gridSize = (n + blockSize - 1) / blockSize;  // Number of blocks

        nQueensKernel<<<gridSize, blockSize>>>(n, d_count);
        hipDeviceSynchronize();  // Wait for the kernel to finish

        int h_count;
        hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

        auto end = std::chrono::high_resolution_clock::now();
        double execution_time = std::chrono::duration<double>(end - start).count();

        std::cout << n << "       " << h_count << "                      " << execution_time << std::endl;

        hipFree(d_count);  // Free device memory
    }

    return 0;
}
